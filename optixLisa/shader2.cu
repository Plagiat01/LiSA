#include "hip/hip_runtime.h"
#include <optix.h>

#include "structs.hh"
#include "random.h"

#include <sutil/vec_math.h>
#include <cuda/helpers.h>

static __forceinline__ __device__ void* unpackPointer( unsigned int i0, unsigned int i1 )
{
    const unsigned long long uptr = static_cast<unsigned long long>( i0 ) << 32 | i1;
    void*           ptr = reinterpret_cast<void*>( uptr );
    return ptr;
}


static __forceinline__ __device__ void  packPointer( void* ptr, unsigned int& i0, unsigned int& i1 )
{
    const unsigned long long uptr = reinterpret_cast<unsigned long long>( ptr );
    i0 = uptr >> 32;
    i1 = uptr & 0x00000000ffffffff;
}

/***** SHADER *****/

extern "C" {
__constant__ Params params;
}

struct Onb
{
  __forceinline__ __device__ Onb(const float3& normal)
  {
    m_normal = normal;

    if( fabs(m_normal.x) > fabs(m_normal.z) )
    {
      m_binormal.x = -m_normal.y;
      m_binormal.y =  m_normal.x;
      m_binormal.z =  0;
    }
    else
    {
      m_binormal.x =  0;
      m_binormal.y = -m_normal.z;
      m_binormal.z =  m_normal.y;
    }

    m_binormal = normalize(m_binormal);
    m_tangent = cross( m_binormal, m_normal );
  }

  __forceinline__ __device__ void inverse_transform(float3& p) const
  {
    p = p.x*m_tangent + p.y*m_binormal + p.z*m_normal;
  }

  float3 m_tangent;
  float3 m_binormal;
  float3 m_normal;
};

static __forceinline__ __device__ void cosine_sample_hemisphere(const float u1, const float u2, float3& p)
{
  // Uniformly sample disk.
  const float r   = sqrtf( u1 );
  const float phi = 2.0f*M_PIf * u2;
  p.x = r * cosf( phi );
  p.y = r * sinf( phi );

  // Project up to hemisphere.
  p.z = sqrtf( fmaxf( 0.0f, 1.0f - p.x*p.x - p.y*p.y ) );
}


struct Intersection
{
  Material material;
  float3 normal;
  float3 xyz;
  float3 mask_color  = make_float3(1.0f);
  float3 accum_color = make_float3(0.0f);
  bool hit = false;
  bool done = false;
  unsigned int seed;
};

extern "C" __device__ float3 shoot_ray_hemisphere(Intersection* intersection) {
  float3 random_dir = make_float3(rnd(intersection->seed),
                                  rnd(intersection->seed),
                                  rnd(intersection->seed));
  
  if (dot(intersection->normal, random_dir) < 0)
      random_dir = -random_dir;
  return random_dir;
}

static __forceinline__ __device__ Intersection* getInter()
{
    const unsigned int u0 = optixGetPayload_0();
    const unsigned int u1 = optixGetPayload_1();
    return reinterpret_cast<Intersection*>(unpackPointer(u0, u1));
}

/**** TRACE FUNCTIONS ****/


static __forceinline__ __device__ void trace_occlusion(OptixTraversableHandle handle,
                                                      float3 ray_origin,
                                                      float3 ray_direction,
                                                      float  tmin,
                                                      float  tmax,
                                                      Intersection* inter)
{
    unsigned int u0, u1;
    packPointer(inter, u0, u1);
    optixTrace(handle,
              ray_origin,
              ray_direction,
              tmin,
              tmax,
              0.0f,                    // rayTime
              OptixVisibilityMask(1),
              OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT,
              RAY_TYPE_OCCLUSION,      // SBT offset
              RAY_TYPE_COUNT,          // SBT stride
              RAY_TYPE_OCCLUSION,      // missSBTIndex
              u0, u1);
}


static __forceinline__ __device__ void trace_radiance(OptixTraversableHandle handle,
                                                      float3 ray_origin,
                                                      float3 ray_direction,
                                                      float  tmin,
                                                      float  tmax,
                                                      Intersection* inter)
{
    unsigned int u0, u1;
    packPointer(inter, u0, u1);
    optixTrace(handle,
              ray_origin,
              ray_direction,
              tmin,
              tmax,
              0.0f,                // rayTime
              OptixVisibilityMask(1),
              OPTIX_RAY_FLAG_NONE,
              RAY_TYPE_RADIANCE,        // SBT offset
              RAY_TYPE_COUNT,           // SBT stride
              RAY_TYPE_RADIANCE,        // missSBTIndex
              u0, u1);
}

extern "C" __device__ void print_color(float3 color) {
  printf("Color: %f %f %f\n", color.x, color.y, color.z);
}


extern "C" __global__ void __raygen__rg() {
  const float2 size = make_float2(params.width, params.height);
  const float3 eye  = params.eye;
  const float3 U    = params.U;
  const float3 V    = params.V;
  const float3 W    = params.W;
  const uint3  idx  = optixGetLaunchIndex();
  const float2  idx2 = make_float2(idx.x, idx.y);

  const int subframe_index = params.subframe_index;
  unsigned int seed        = tea<4>(idx.y*size.x + idx.x, subframe_index);

  const int samples_per_launch = params.samples_per_launch;
  const int nb_bounces = 3; 

  Intersection intersection;
  intersection.seed        = seed;

  for (int i = 0; i < samples_per_launch; i++) {
    
    /* Builds ray direction/origin */
    const float2 antialiasing_jitter = make_float2(rnd(seed), rnd(seed));
    const float3 d                   = make_float3((2.0f * idx2 + antialiasing_jitter) / size - 1.0f, 1.0f);
    float3 ray_direction             = normalize(d.x*U + d.y*V + W);
    float3 ray_origin                = eye;

    for (int j = 0; j < nb_bounces; j++) {
      intersection.done = false;

      trace_radiance(params.handle,
                    ray_origin,
                    ray_direction,
                    0.01f,
                    1e16f,
                    &intersection);

      if (intersection.done) break;

      ray_origin    = intersection.xyz;
      ray_direction = shoot_ray_hemisphere(&intersection);
    }
  }
  float3 accum_color = intersection.accum_color;
  const uint3 launch_index       = optixGetLaunchIndex();
  const unsigned int image_index = launch_index.y * params.width + launch_index.x;
  accum_color                    = accum_color / static_cast<float>(samples_per_launch);

  if( subframe_index > 0 ) {
      const float a                  = 1.0f / static_cast<float>(subframe_index + 1);
      const float3 accum_color_prev = make_float3(params.accum_buffer[image_index]);
      accum_color = lerp(accum_color_prev, accum_color, a);
  }
  params.accum_buffer[ image_index ] = make_float4(accum_color, 1.0f);
  params.frame_buffer[ image_index ] = make_color (accum_color);
}


/**** OCCLUSION ****/

extern "C" __global__ void __miss__occlusion() {
  Intersection* intersection = getInter();
  intersection->hit = false;
}

extern "C" __global__ void __closesthit__occlusion() {
  HitGroupData* rt_data = reinterpret_cast<HitGroupData*>(optixGetSbtDataPointer());
  if (rt_data->material.emit) {
    Intersection* intersection = getInter();
    intersection->material = rt_data->material;
    intersection->hit = true;
  }
}


/**** RADIANCE ****/

extern "C" __global__ void __miss__radiance() {
    MissData* rt_data  = reinterpret_cast<MissData*>(optixGetSbtDataPointer());
    Intersection* intersection = getInter();

    intersection->done        = true;
    intersection->accum_color += make_float3(rt_data->bg_color) * intersection->mask_color;
}

extern "C" __device__ float3 shoot_ray_to_light(Intersection* intersection) {
  const unsigned int count = 7u;
  for (int i = 0; i < count; i++) {
    const float z1 = rnd(intersection->seed);
    const float z2 = rnd(intersection->seed);

    float3 dir;
    cosine_sample_hemisphere( z1, z2, dir );
    Onb onb(intersection->normal);
    onb.inverse_transform(dir);

    trace_occlusion(params.handle, intersection->xyz, dir, 0.01f, 1e16f, intersection);

    if (intersection->hit) {
      const float d = clamp(dot(intersection->normal, dir), 0.0f, 1.0f);
      return d * intersection->material.emission_color;
    }
  }
  return make_float3(0.0f);
}

extern "C" __global__ void __closesthit__radiance() {

  HitGroupData* rt_data = reinterpret_cast<HitGroupData*>(optixGetSbtDataPointer());
  Intersection* intersection = getInter();
  
  if (rt_data->material.emit) {
    intersection->accum_color += rt_data->material.emission_color * intersection->mask_color;
    intersection->done = true;
  } else {
    intersection->mask_color *= rt_data->material.diffuse_color;

    const int    prim_idx        = optixGetPrimitiveIndex();
    const float3 ray_dir         = optixGetWorldRayDirection();
    const int    vert_idx_offset = prim_idx*3;

    const float3 v1      = make_float3(rt_data->vertices[vert_idx_offset + 0]);
    const float3 v2      = make_float3(rt_data->vertices[vert_idx_offset + 1]);
    const float3 v3      = make_float3(rt_data->vertices[vert_idx_offset + 2]);
    const float3 N_0     = normalize(cross(v2-v1, v3-v1));
    intersection->xyz    = optixGetWorldRayOrigin() + optixGetRayTmax() * ray_dir;
    intersection->normal = faceforward(N_0, -ray_dir, N_0);

    intersection->accum_color += shoot_ray_to_light(intersection) * intersection->mask_color;
  }
}